#include "hip/hip_runtime.h"
#include "rosalind.h"

#include "hip/hip_runtime.h"
#include ""
// Problem 6: Counting Point Mutations

__global__ void hammkernel(const char* first, const char* second,  int* hamming) {
	if (first[threadIdx.x] == second[threadIdx.x])
		atomicAdd(hamming, 1);
}

hipError_t hammcuda(const char* first, const char* second, int size, int* hamming) {
	char* devfirst = 0;
	char* devsecond = 0;
	int* devhamming = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't find CUDA-compatible device.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devfirst, size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for input on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devsecond, size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for input on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devhamming, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for output on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(devfirst, first, size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy input to GPU: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(devsecond, second, size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy input to GPU: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hammkernel <<<1, size>>> (devfirst, devsecond, devhamming);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't execute kernel to calculate hamming distance: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't synchronize threads for hammkernel: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(hamming, devhamming, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy output to host device.\n");
		goto Error;
	}

Error:
	hipFree(devfirst);
	hipFree(devsecond);
	hipFree(devhamming);

	return cudaStatus;
}

