#include "hip/hip_runtime.h"
#include "rosalind.h"

#include "hip/hip_runtime.h"
#include ""
// Problem 2: Transcribing DNA into RNA

__global__ void revckernel(const char* in, char* out)
{
	int oindex = blockDim.x - threadIdx.x;
	switch (in[threadIdx.x]) {
	case 'A':
		out[oindex] = 'T';
		break;
	case 'C':
		out[oindex] = 'G';
		break;
	case 'G':
		out[oindex] = 'C';
		break;
	case 'T':
		out[oindex] = 'A';
		break;
	default:
		break;
	}
}

hipError_t dnacomplement(const char* in, char* out, unsigned int size)
{
	char* devin = 0;
	char* devout = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't find CUDA-compatible device.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devin, size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for input on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devout, size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for output on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(devin, in, size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy input to GPU: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	revckernel <<<1, size>>> (devin, devout);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't execute ntcountkernel: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't synchronize threads for ntcountkernel: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	/* We do a bunch of terrible string hacks here because the null character keeps getting copied to the beginning of the string.
	I should come up with a more elegant solution, in case this comes up in future problems. */
	cudaStatus = hipMemcpy(out, devout + 1, (size - 1) * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy output to host device.\n");
		goto Error;
	}
	switch (in[0]) {
	case 'A':
		out[size - 1] = 'T';
		break;
	case 'C':
		out[size - 1] = 'G';
		break;
	case 'G':
		out[size - 1] = 'C';
		break;
	case 'T':
		out[size - 1] = 'A';
		break;
	default:
		break;
	}
	out[size] = '\0';

Error:
	hipFree(devin);
	hipFree(devout);

	return cudaStatus;
}