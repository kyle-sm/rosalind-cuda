#include "hip/hip_runtime.h"
#include "rosalind.h"

#include "hip/hip_runtime.h"
#include ""
// Problem 2: Transcribing DNA into RNA

__global__ void transcribekernel(const char* in, char* out)
{
	if (in[threadIdx.x] == 'T')
		out[threadIdx.x] = 'U';
	else
		out[threadIdx.x] = in[threadIdx.x];
}

hipError_t rnatranscribe(const char* in, char* out, unsigned int size)
{
	char* devin = 0;
	char* devout = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't find CUDA-compatible device.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devin, size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for input on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devout, size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for output on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(devin, in, size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy input to GPU: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	transcribekernel <<<1, size>>> (devin, devout);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't execute ntcountkernel: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't synchronize threads for ntcountkernel: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(out, devout, size * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy output to host device.\n");
		goto Error;
	}

Error:
	hipFree(devin);
	hipFree(devout);

	return cudaStatus;
}