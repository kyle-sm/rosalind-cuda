#include "hip/hip_runtime.h"
#include "rosalind.h"

#include "hip/hip_runtime.h"
#include ""
// Problem 1: Counting DNA Nucleotides

__global__ void ntcountkernel(const char* in, int* out)
{
	switch (in[threadIdx.x]) {
	case 'A':
		atomicAdd(&out[0], 1);
		break;
	case 'C':
		atomicAdd(&out[1], 1);
		break;
	case 'G':
		atomicAdd(&out[2], 1);
		break;
	case 'T':
		atomicAdd(&out[3], 1);
		break;
	default:
		break;
	}
}

hipError_t ntcount(const char* in, int* out, unsigned int size)
{
	char* devin = 0;
	int* devout = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't find CUDA-compatible device.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devin, size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for input on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devout, 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't allocate memory for output on GPU.\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(devin, in, size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy input to GPU: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	ntcountkernel <<<1, size>>> (devin, devout);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't execute ntcountkernel: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't synchronize threads for ntcountkernel: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(out, devout, 4 * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Couldn't copy output to host device.\n");
		goto Error;
	}


Error:
	hipFree(devin);
	hipFree(devout);

	return cudaStatus;
}